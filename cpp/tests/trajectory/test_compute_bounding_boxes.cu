#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/column_utilities.hpp>

#include "tests/trajectory/trajectory_utilities.cuh"

struct TrajectoryDistanceSpeedTest : public cudf::test::BaseFixture {};

constexpr cudf::size_type size{1000};

TEST_F(TrajectoryDistanceSpeedTest,
       ComputeBoundingBoxesForThreeTrajectories) {
  auto sorted = cuspatial::test::make_test_trajectories_table(size);
  auto id = sorted->get_column(0);
  auto ts = sorted->get_column(1);
  auto xs = sorted->get_column(2);
  auto ys = sorted->get_column(3);

  auto offsets = cuspatial::experimental::compute_trajectory_offsets(id, this->mr());

  auto bounding_boxes =
      cuspatial::experimental::compute_bounding_boxes(xs, ys, *offsets, this->mr());

  auto h_xs = cudf::test::to_host<double>(xs).first;
  auto h_ys = cudf::test::to_host<double>(ys).first;
  auto h_ts = cudf::test::to_host<cudf::timestamp_ms>(ts).first;
  auto h_offsets = cudf::test::to_host<int32_t>(*offsets).first;

  std::vector<double> bbox_x1(h_offsets.size());
  std::vector<double> bbox_y1(h_offsets.size());
  std::vector<double> bbox_x2(h_offsets.size());
  std::vector<double> bbox_y2(h_offsets.size());

  // compute expected bounding boxes
  for (size_t tid = 0; tid < h_offsets.size(); ++tid) {
    auto end = h_offsets[tid] - 1;
    auto idx = tid == 0 ? 0 : h_offsets[tid - 1];

    auto x1 = h_xs[idx];
    auto y1 = h_ys[idx];
    auto x2 = h_xs[idx];
    auto y2 = h_ys[idx];

    for (int32_t i = idx; ++i < end;) {
      x1 = std::min(x1, h_xs[i]);
      y1 = std::min(y1, h_ys[i]);
      x2 = std::max(x2, h_xs[i]);
      y2 = std::max(y2, h_ys[i]);
    }

    bbox_x1[tid] = x1;
    bbox_y1[tid] = y1;
    bbox_x2[tid] = x2;
    bbox_y2[tid] = y2;
  }

  cudf::test::expect_columns_equivalent(
      bounding_boxes->get_column(0),
      cudf::test::fixed_width_column_wrapper<double>(bbox_x1.begin(),
                                                     bbox_x1.end()));
  cudf::test::expect_columns_equivalent(
      bounding_boxes->get_column(1),
      cudf::test::fixed_width_column_wrapper<double>(bbox_y1.begin(),
                                                     bbox_y1.end()));
  cudf::test::expect_columns_equivalent(
      bounding_boxes->get_column(2),
      cudf::test::fixed_width_column_wrapper<double>(bbox_x2.begin(),
                                                     bbox_x2.end()));
  cudf::test::expect_columns_equivalent(
      bounding_boxes->get_column(3),
      cudf::test::fixed_width_column_wrapper<double>(bbox_y2.begin(),
                                                     bbox_y2.end()));
}
