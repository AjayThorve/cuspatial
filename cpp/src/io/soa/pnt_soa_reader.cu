#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <utilities/error_utils.hpp>
#include <rmm/rmm.h>
#include <cuspatial/soa_readers.hpp>
#include <cuspatial/shared_util.h>

namespace cuSpatial
{
	/**
	 * @Brief read lon/lat from file into two columns; data type is fixed to double (GDF_FLOAT64)
	 * see soa_readers.hpp
	*
	*/
	void read_pnt_lonlat_soa(const char *pnt_fn,gdf_column& pnt_lon,gdf_column& pnt_lat)                                   
	{

		double * p_lon=NULL, *p_lat=NULL;
		int num_p=read_point_ll<double>(pnt_fn,p_lon,p_lat);
		
 		pnt_lon.dtype= GDF_FLOAT64;
 		pnt_lon.col_name=(char *)malloc(strlen("lon")+ 1);
		strcpy(pnt_lon.col_name,"lon");
		RMM_TRY( RMM_ALLOC(&pnt_lon.data, num_p * sizeof(double), 0) );
		hipMemcpy(pnt_lon.data, p_lon,num_p * sizeof(double) , hipMemcpyHostToDevice);		
		pnt_lon.size=num_p;
		pnt_lon.valid=nullptr;
		pnt_lon.null_count=0;		
		delete[] p_lon;

 		pnt_lat.dtype= GDF_FLOAT64;
 		pnt_lat.col_name=(char *)malloc(strlen("lat")+ 1);
		strcpy(pnt_lat.col_name,"lat");
		pnt_lat.data=NULL;
		RMM_TRY( RMM_ALLOC(&pnt_lat.data, num_p * sizeof(double), 0) );
		hipMemcpy(pnt_lat.data, p_lat,num_p * sizeof(double) , hipMemcpyHostToDevice);		
		pnt_lat.size=num_p;
		pnt_lat.valid=nullptr;
		pnt_lat.null_count=0;
		delete[] p_lat;
	}
	
	/**
	 * @Brief read x/y from file into two columns; data type is fixed to double (GDF_FLOAT64)
	*
	*/
	void read_pnt_xy_soa(const char *pnt_fn,gdf_column& pnt_x,gdf_column& pnt_y)                                   
	{

		double * p_x=NULL, *p_y=NULL;
		int num_p=read_point_xy<double>(pnt_fn,p_x,p_y);
		
 		pnt_x.dtype= GDF_FLOAT64;
 		pnt_x.col_name=(char *)malloc(strlen("x")+ 1);
		strcpy(pnt_x.col_name,"x");
		RMM_TRY( RMM_ALLOC(&pnt_x.data, num_p * sizeof(double), 0) );
		hipMemcpy(pnt_x.data, p_x,num_p * sizeof(double) , hipMemcpyHostToDevice);		
		pnt_x.size=num_p;
		pnt_x.valid=nullptr;
		pnt_x.null_count=0;		
		delete[] p_x;

 		pnt_y.dtype= GDF_FLOAT64;
 		pnt_y.col_name=(char *)malloc(strlen("y")+ 1);
		strcpy(pnt_y.col_name,"y");
		pnt_y.data=NULL;
		RMM_TRY( RMM_ALLOC(&pnt_y.data, num_p * sizeof(double), 0) );
		hipMemcpy(pnt_y.data, p_y,num_p * sizeof(double) , hipMemcpyHostToDevice);		
		pnt_y.size=num_p;
		pnt_y.valid=nullptr;
		pnt_y.null_count=0;
		delete[] p_y;
	}	
}