/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <rmm/rmm.h>
#include <cudf/types.h>
#include <utilities/error_utils.hpp>
#include <cuspatial/soa_readers.hpp>
#include <utility/utility.hpp>

namespace cuspatial
{
	/**
	 * @brief read poygon data from file in SoA format; data type of vertices is fixed to double (GDF_FLOAT64)
	 * see soa_readers.hpp
	*/	
	
	void read_polygon_soa(const char *filename,gdf_column* ply_fpos, gdf_column* ply_rpos,
		gdf_column* ply_x,gdf_column* ply_y)
	{
	        CUDF_EXPECTS(ply_fpos!=nullptr && ply_rpos!=nullptr && ply_x!=nullptr && ply_y!=nullptr,
	        	"none of the polygon offset/ring offset/x coorinate/y coordinate column can be null ");
	        
	        memset(ply_fpos,0,sizeof(gdf_column));
	        memset(ply_rpos,0,sizeof(gdf_column));
	        memset(ply_x,0,sizeof(gdf_column));
	        memset(ply_y,0,sizeof(gdf_column));
	
	        struct polygons<double> pm;
	        read_polygon_soa<double>(filename,&pm);	        
	        if(pm.num_feature<=0) return;
	        
  		ply_fpos->dtype=GDF_INT32;
  		ply_fpos->col_name=(char *)malloc(strlen("f_pos")+ 1);
		strcpy(ply_fpos->col_name,"f_pos");
		ply_fpos->data=nullptr;
		RMM_TRY( RMM_ALLOC(&(ply_fpos->data), pm.num_feature * sizeof(uint32_t), 0) );
		hipMemcpy(ply_fpos->data, pm.feature_length,pm.num_feature * sizeof(uint32_t) , hipMemcpyHostToDevice);
		thrust::device_ptr<uint32_t> d_pfp_ptr=thrust::device_pointer_cast((uint32_t *)(ply_fpos->data));
		//prefix-sum: len to pos
		thrust::inclusive_scan(d_pfp_ptr,d_pfp_ptr+pm.num_feature,d_pfp_ptr);
		ply_fpos->size=pm.num_feature;
		ply_fpos->valid=nullptr;
		ply_fpos->null_count=0;
		delete[] pm.feature_length;

 		ply_rpos->dtype=GDF_INT32;
 		ply_rpos->col_name=(char *)malloc(strlen("r_pos")+ 1);
		strcpy(ply_rpos->col_name,"r_pos");
		ply_rpos->data=nullptr;
		RMM_TRY( RMM_ALLOC(&(ply_rpos->data), pm.num_ring * sizeof(uint32_t), 0) );
		hipMemcpy(ply_rpos->data, pm.ring_length,pm.num_ring * sizeof(uint32_t) , hipMemcpyHostToDevice);
		thrust::device_ptr<uint32_t> d_prp_ptr=thrust::device_pointer_cast((uint32_t *)(ply_rpos->data));
		//prefix-sum: len to pos
		thrust::inclusive_scan(d_prp_ptr,d_prp_ptr+pm.num_ring,d_prp_ptr);
		ply_rpos->size=pm.num_ring;
		ply_rpos->valid=nullptr;
		ply_rpos->null_count=0;
		delete[] pm.ring_length;

 		ply_x->dtype= GDF_FLOAT64;
 		ply_x->col_name=(char *)malloc(strlen("x")+ 1);
		strcpy(ply_x->col_name,"x");
		RMM_TRY( RMM_ALLOC(&(ply_x->data), pm.num_vertex * sizeof(double), 0) );
		hipMemcpy(ply_x->data, pm.x,pm.num_vertex * sizeof(double) , hipMemcpyHostToDevice);		
		ply_x->size=pm.num_vertex;
		ply_x->valid=nullptr;
		ply_x->null_count=0;		
		delete[] pm.x;

 		ply_y->dtype= GDF_FLOAT64;
 		ply_y->col_name=(char *)malloc(strlen("y")+ 1);
		strcpy(ply_y->col_name,"y");
		ply_y->data=nullptr;
		RMM_TRY( RMM_ALLOC(&(ply_y->data), pm.num_vertex * sizeof(double), 0) );
		hipMemcpy(ply_y->data, pm.y,pm.num_vertex * sizeof(double) , hipMemcpyHostToDevice);		
		ply_y->size=pm.num_vertex;
		ply_y->valid=nullptr;
		ply_y->null_count=0;
		delete[] pm.y;
		
		delete[] pm.group_length;
	}//read_polygon_soa
}// namespace cuspatial
