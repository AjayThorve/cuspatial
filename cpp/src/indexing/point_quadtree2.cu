#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/table/table.hpp>
#include <cuspatial/detail/point_quadtree.hpp>
#include <cuspatial/error.hpp>
#include <memory>
#include <tuple>
#include <type_traits>
#include <utility>
#include <vector>

#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include "utility/z_order.cuh"

/*
 * quadtree indexing on points using the bottom-up algorithm described at ref.
 * http://www.adms-conf.org/2019-camera-ready/zhang_adms19.pdf
 * extra care on minmizing peak device memory usage by deallocating memory as
 * early as possible
 */

namespace {

template <typename Vector>
inline auto shrink_vector(Vector &v, cudf::size_type size) {
  v.resize(size, 0);
  v.shrink_to_fit();
  return v;
}

template <typename... Ts>
inline auto make_zip_iterator(Ts... its) {
  return thrust::make_zip_iterator(
      thrust::make_tuple(std::forward<Ts>(its)...));
}

template <typename T>
inline std::unique_ptr<cudf::column> make_fixed_width_column(
    cudf::size_type size, hipStream_t stream = 0,
    rmm::mr::device_memory_resource *mr = rmm::mr::get_default_resource()) {
  return cudf::make_fixed_width_column(
      cudf::data_type{cudf::experimental::type_to_id<T>()}, size,
      cudf::mask_state::UNALLOCATED, stream, mr);
}

template <typename T, typename PointToKeyFunc>
inline std::unique_ptr<cudf::column> compute_z_keys(
    cudf::mutable_column_view &x, cudf::mutable_column_view &y,
    hipStream_t stream, PointToKeyFunc func) {
  auto policy = rmm::exec_policy(stream);
  auto points = make_zip_iterator(x.begin<T>(), y.begin<T>());
  auto keys = make_fixed_width_column<int32_t>(x.size(), stream);

  // Compute Morton codes (z-order) for each point
  thrust::transform(policy->on(stream), points, points + x.size(),
                    keys->mutable_view().begin<uint32_t>(), func);
  // Sort the points and codes
  thrust::sort_by_key(policy->on(stream),
                      keys->mutable_view().begin<uint32_t>(),
                      keys->mutable_view().end<uint32_t>(), points);

  return keys;
}

template <typename InputIterator1, typename InputIterator2,
          typename OutputIterator1, typename OutputIterator2,
          typename BinaryPred>
inline cudf::size_type compute_full_quads(
    InputIterator1 keys_begin, InputIterator1 keys_end, InputIterator2 vals_in,
    OutputIterator1 keys_out, OutputIterator2 vals_out, BinaryPred binary_op,
    hipStream_t stream) {
  auto policy = rmm::exec_policy(stream);
  auto result = thrust::reduce_by_key(policy->on(stream), keys_begin, keys_end,
                                      vals_in, keys_out, vals_out,
                                      thrust::equal_to<uint32_t>(), binary_op);
  return thrust::distance(keys_out, result.first);
}

template <typename T>
struct tuple_sum {
  inline __device__ thrust::tuple<T, T> operator()(
      thrust::tuple<T, T> const &a, thrust::tuple<T, T> const &b) {
    return thrust::make_tuple(thrust::get<0>(a) + thrust::get<0>(b),
                              thrust::get<1>(a) + thrust::get<1>(b));
  }
};

template <typename KeysIterator, typename ValsIterator>
inline std::tuple<cudf::size_type, cudf::size_type, std::vector<uint32_t>,
                  std::vector<uint32_t>>
compute_full_levels(cudf::size_type const num_levels,
                    cudf::size_type const num_top_quads,
                    KeysIterator keys_begin,
                    ValsIterator quad_point_count_begin,
                    ValsIterator quad_child_count_begin, hipStream_t stream) {
  // begin/end offsets
  cudf::size_type begin{0};
  cudf::size_type end{num_top_quads};
  std::vector<uint32_t> b_pos(num_levels);
  std::vector<uint32_t> e_pos(num_levels);

  // iterator for the parent level's quad node keys
  auto parent_keys = thrust::make_transform_iterator(
      keys_begin, [] __device__(uint32_t const child) { return (child >> 2); });

  // iterator for the current level's quad node point and child counts
  auto child_nodes =
      make_zip_iterator(quad_point_count_begin, quad_child_count_begin);

  // iterator for the current level's initial values
  auto child_values = make_zip_iterator(
      quad_point_count_begin, thrust::make_constant_iterator<uint32_t>(1));

  for (cudf::size_type level = num_levels - 1; level >= 0; --level) {
    auto range = compute_full_quads(
        parent_keys + begin, parent_keys + end, child_values + begin,
        keys_begin + end, child_nodes + end, tuple_sum<uint32_t>{}, stream);
    e_pos[level] = end;
    b_pos[level] = begin;
    begin = end;
    end += range;
  }

  return std::make_tuple(
      // count the number of parent nodes (excluding leaf nodes)
      end - num_top_quads - 1,  //
      end, b_pos, e_pos);
}

inline std::tuple<rmm::device_vector<uint32_t>, rmm::device_vector<uint32_t>,
                  rmm::device_vector<uint32_t>, rmm::device_vector<int8_t>>
reverse_tree_levels(rmm::device_vector<uint32_t> const &quad_keys_in,
                    rmm::device_vector<uint32_t> const &quad_point_count_in,
                    rmm::device_vector<uint32_t> const &quad_child_count_in,
                    std::vector<uint32_t> b_pos, std::vector<uint32_t> e_pos,
                    cudf::size_type const num_levels, hipStream_t stream) {
  auto policy = rmm::exec_policy(stream);
  rmm::device_vector<uint32_t> quad_keys(quad_keys_in.size());
  rmm::device_vector<int8_t> quad_level(quad_keys_in.size());
  rmm::device_vector<uint32_t> quad_point_count(quad_point_count_in.size());
  rmm::device_vector<uint32_t> quad_child_count(quad_child_count_in.size());
  cudf::size_type offset{0};

  for (cudf::size_type level{0}; level < num_levels; ++level) {
    cudf::size_type end = e_pos[level];
    cudf::size_type begin = b_pos[level];
    cudf::size_type range = e_pos[level] - b_pos[level];
    thrust::fill(policy->on(stream), quad_level.begin() + offset,
                 quad_level.begin() + offset + range, level);
    thrust::copy(policy->on(stream), quad_keys_in.begin() + begin,
                 quad_keys_in.begin() + end, quad_keys.begin() + offset);
    thrust::copy(policy->on(stream), quad_point_count_in.begin() + begin,
                 quad_point_count_in.begin() + end,
                 quad_point_count.begin() + offset);
    thrust::copy(policy->on(stream), quad_child_count_in.begin() + begin,
                 quad_child_count_in.begin() + end,
                 quad_child_count.begin() + offset);
    // thrust::reduce(policy->on(stream), quad_point_count_in.begin() + begin,
    //                quad_point_count_in.begin() + end);
    offset += range;
  }

  // Shrink vectors' underlying device allocations to reduce peak memory usage
  quad_keys.shrink_to_fit();
  quad_point_count.shrink_to_fit();
  quad_child_count.shrink_to_fit();
  quad_level.shrink_to_fit();

  return std::make_tuple(quad_keys, quad_point_count, quad_child_count,
                         quad_level);
}

inline rmm::device_vector<uint32_t> compute_parent_positions(
    rmm::device_vector<uint32_t> const &quad_child_count,
    cudf::size_type const num_parent_nodes,
    cudf::size_type const num_child_nodes, hipStream_t stream) {
  // compute parent node start positions
  auto policy = rmm::exec_policy(stream);
  // wrap in an IEFE so `position_map` is freed on return
  auto parent_pos = [&]() {
    rmm::device_vector<uint32_t> position_map(num_parent_nodes);
    // line 1 of algorithm in Fig. 5 in ref.
    thrust::exclusive_scan(policy->on(stream), quad_child_count.begin(),
                           quad_child_count.begin() + num_parent_nodes,
                           position_map.begin());
    // line 2 of algorithm in Fig. 5 in ref.
    rmm::device_vector<uint32_t> parent_pos(num_child_nodes);
    thrust::scatter(policy->on(stream), thrust::make_counting_iterator(0),
                    thrust::make_counting_iterator(0) + num_parent_nodes,
                    position_map.begin(), parent_pos.begin());
    return parent_pos;
  }();

  // line 3 of algorithm in Fig. 5 in ref.
  thrust::inclusive_scan(policy->on(stream), parent_pos.begin(),
                         parent_pos.begin() + num_child_nodes,
                         parent_pos.begin(), thrust::maximum<uint32_t>());

  return parent_pos;
}

inline std::pair<uint32_t, uint32_t> remove_unqualified_quads(
    rmm::device_vector<uint32_t> &quad_keys,
    rmm::device_vector<uint32_t> &quad_point_count,
    rmm::device_vector<uint32_t> &quad_child_count,
    rmm::device_vector<int8_t> &quad_level,
    cudf::size_type const num_parent_nodes,
    cudf::size_type const num_child_nodes, cudf::size_type const min_size,
    cudf::size_type const level_1_size, hipStream_t stream) {
  // remove invalid nodes, return number of valid nodes left
  auto policy = rmm::exec_policy(stream);
  // compute parent node start positions
  auto parent_positions = compute_parent_positions(
      quad_child_count, num_parent_nodes, num_child_nodes, stream);
  auto parent_point_counts = thrust::make_permutation_iterator(
      quad_point_count.begin(), parent_positions.begin());

  // Count the number of nodes whose children have fewer points than `min_size`.
  // Start counting nodes at level 2, since children of the root node should not
  // be discarded.
  auto num_invalid_parent_nodes = thrust::count_if(
      policy->on(stream), parent_point_counts,
      parent_point_counts + (num_parent_nodes - level_1_size),
      // i.e. quad_point_count[parent_pos] <= min_size
      [min_size] __device__(auto const n) { return n <= min_size; });

  // line 4 of algorithm in Fig. 5 in ref.
  // revision to line 4: copy unnecessary if using permutation_iterator stencil

  // Remove quad nodes fewer points than min_size.
  // Start counting nodes at level 2, since children of the root node should not
  // be discarded.
  // line 5 of algorithm in Fig. 5 in ref.
  auto tree = make_zip_iterator(quad_keys.begin() + level_1_size,
                                quad_point_count.begin() + level_1_size,
                                quad_child_count.begin() + level_1_size,
                                quad_level.begin() + level_1_size);

  auto last_valid = thrust::remove_if(
      policy->on(stream), tree, tree + num_child_nodes, parent_point_counts,
      // i.e. quad_point_count[parent_pos] <= min_size
      [min_size] __device__(auto const n) { return n <= min_size; });

  // add the number of level 1 nodes back in to num_valid_nodes
  auto num_valid_nodes = thrust::distance(tree, last_valid) + level_1_size;

  return std::make_pair(num_invalid_parent_nodes, num_valid_nodes);
}

inline std::unique_ptr<cudf::column> construct_non_leaf_indicator(
    rmm::device_vector<uint32_t> &quad_point_count,
    cudf::size_type const num_parent_nodes,
    cudf::size_type const num_valid_nodes, cudf::size_type const min_size,
    rmm::mr::device_memory_resource *mr, hipStream_t stream) {
  //
  auto policy = rmm::exec_policy(stream);
  // Construct the indicator output column
  auto indicator = make_fixed_width_column<bool>(num_valid_nodes, stream, mr);

  // line 6 of algorithm in Fig. 5 in ref.
  thrust::transform(policy->on(stream), quad_point_count.begin(),
                    quad_point_count.begin() + num_parent_nodes,
                    indicator->mutable_view().begin<bool>(),
                    thrust::placeholders::_1 > min_size);

  // line 7 of algorithm in Fig. 5 in ref.
  thrust::replace_if(policy->on(stream), quad_point_count.begin(),
                     quad_point_count.begin() + num_parent_nodes,
                     indicator->view().begin<bool>(), thrust::placeholders::_1,
                     0);

  if (num_valid_nodes > num_parent_nodes) {
    // zero-fill the rest of the indicator column because
    // device_memory_resources aren't required to initialize allocations
    thrust::fill(policy->on(stream),
                 indicator->mutable_view().begin<bool>() + num_parent_nodes,
                 indicator->mutable_view().end<bool>(), 0);
  }

  return indicator;
}

inline rmm::device_vector<uint32_t> compute_leaf_positions(
    cudf::column_view const &indicator, cudf::size_type const num_valid_nodes,
    hipStream_t stream) {
  auto policy = rmm::exec_policy(stream);
  rmm::device_vector<uint32_t> leaf_pos(num_valid_nodes);
  auto result = thrust::copy_if(
      policy->on(stream), thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(0) + num_valid_nodes,
      indicator.begin<bool>(), leaf_pos.begin(), !thrust::placeholders::_1);
  // Shrink leaf_pos's underlying device allocation
  return shrink_vector(leaf_pos, thrust::distance(leaf_pos.begin(), result));
}

inline rmm::device_vector<uint32_t> flatten_point_keys(
    rmm::device_vector<uint32_t> const &quad_keys,
    rmm::device_vector<int8_t> const &quad_level,
    cudf::column_view const &indicator, cudf::size_type const num_valid_nodes,
    cudf::size_type const num_levels, hipStream_t stream) {
  rmm::device_vector<uint32_t> flattened_keys(num_valid_nodes);
  auto policy = rmm::exec_policy(stream);
  auto keys_and_levels = make_zip_iterator(
      quad_keys.begin(), quad_level.begin(), indicator.begin<bool>());
  thrust::transform(
      policy->on(stream), keys_and_levels, keys_and_levels + num_valid_nodes,
      flattened_keys.begin(), [M = num_levels] __device__(auto const &val) {
        bool is_node{false};
        uint32_t key{}, level{};
        thrust::tie(key, level, is_node) = val;
        return is_node ? 0xFFFFFFFF : (key << (2 * (M - 1 - level)));
      });
  flattened_keys.shrink_to_fit();
  return flattened_keys;
}

inline rmm::device_vector<uint32_t> compute_flattened_first_point_positions(
    rmm::device_vector<uint32_t> const &quad_keys,
    rmm::device_vector<int8_t> const &quad_level,
    rmm::device_vector<uint32_t> &quad_point_count,
    cudf::column_view const &indicator, cudf::size_type const num_valid_nodes,
    cudf::size_type const num_levels, hipStream_t stream) {
  //
  // Adjust quad_point_count and quad_point_pos based on the last level's
  // z-order keys
  //
  auto policy = rmm::exec_policy(stream);

  rmm::device_vector<uint32_t> initial_sort_indices{};
  rmm::device_vector<uint32_t> quad_point_count_tmp{};
  // Sort initial indices and temporary point counts by the flattened keys
  std::tie(initial_sort_indices, quad_point_count_tmp) = [&]() {
    auto flattened_keys = flatten_point_keys(
        quad_keys, quad_level, indicator, num_valid_nodes, num_levels, stream);

    rmm::device_vector<uint32_t> initial_sort_indices(num_valid_nodes);
    thrust::sequence(policy->on(stream), initial_sort_indices.begin(),
                     initial_sort_indices.end());

    rmm::device_vector<uint32_t> quad_point_count_tmp(num_valid_nodes);
    thrust::copy(policy->on(stream), quad_point_count.begin(),
                 quad_point_count.end(), quad_point_count_tmp.begin());

    // sort indices and temporary point counts
    thrust::stable_sort_by_key(policy->on(stream), flattened_keys.begin(),
                               flattened_keys.end(),
                               make_zip_iterator(initial_sort_indices.begin(),
                                                 quad_point_count_tmp.begin()));

    thrust::remove_if(policy->on(stream), quad_point_count_tmp.begin(),
                      quad_point_count_tmp.begin() + num_valid_nodes,
                      quad_point_count_tmp.begin(),
                      thrust::placeholders::_1 == 0);

    initial_sort_indices.shrink_to_fit();
    quad_point_count_tmp.shrink_to_fit();

    return std::make_pair(initial_sort_indices, quad_point_count_tmp);
  }();

  auto leaf_pos = compute_leaf_positions(indicator, num_valid_nodes, stream);

  // Shrink the vector's underlying device allocations.
  // Only the first `num_leaf_nodes` are needed after removal, since
  // copy_if and remove_if should remove the same number of elements.
  shrink_vector(quad_point_count_tmp, leaf_pos.size());
  shrink_vector(initial_sort_indices, leaf_pos.size());

  rmm::device_vector<uint32_t> quad_point_f_pos_tmp(leaf_pos.size());

  thrust::exclusive_scan(policy->on(stream), quad_point_count_tmp.begin(),
                         quad_point_count_tmp.end(),
                         quad_point_f_pos_tmp.begin());

  auto count_and_f_pos = make_zip_iterator(quad_point_count_tmp.begin(),
                                           quad_point_f_pos_tmp.begin());

  thrust::stable_sort_by_key(policy->on(stream), initial_sort_indices.begin(),
                             initial_sort_indices.end(), count_and_f_pos);

  rmm::device_vector<uint32_t> quad_point_f_pos(num_valid_nodes);

  thrust::scatter(
      policy->on(stream), count_and_f_pos, count_and_f_pos + leaf_pos.size(),
      leaf_pos.begin(),
      make_zip_iterator(quad_point_count.begin(), quad_point_f_pos.begin()));

  quad_point_f_pos.shrink_to_fit();

  return quad_point_f_pos;
}

template <typename TypeOut, typename TypeIn>
inline std::unique_ptr<cudf::column> copy_if_else(
    rmm::device_vector<TypeIn> const &lhs,
    rmm::device_vector<TypeIn> const &rhs, cudf::column_view const &mask,
    cudf::size_type const size, rmm::mr::device_memory_resource *mr,
    hipStream_t stream) {
  // for each value in `mask` copy from `lhs` if true, else `rhs`
  auto policy = rmm::exec_policy(stream);
  auto output = make_fixed_width_column<TypeOut>(size, stream, mr);
  auto iter = make_zip_iterator(mask.begin<bool>(), lhs.begin(), rhs.begin());

  thrust::transform(policy->on(stream), iter, iter + size,
                    output->mutable_view().template begin<TypeIn>(),
                    // return bool ? lhs : rhs
                    [] __device__(auto const &t) {
                      return thrust::get<0>(t) ? thrust::get<1>(t)
                                               : thrust::get<2>(t);
                    });

  return output;
}

inline std::unique_ptr<cudf::experimental::table> make_full_quadtree(
    rmm::device_vector<uint32_t> &quad_keys,
    rmm::device_vector<uint32_t> &quad_point_count,
    rmm::device_vector<uint32_t> &quad_child_count,
    rmm::device_vector<int8_t> &quad_level, cudf::size_type num_parent_nodes,
    cudf::size_type const quad_tree_size, cudf::size_type const num_levels,
    cudf::size_type const min_size, cudf::size_type const level_1_size,
    rmm::mr::device_memory_resource *mr, hipStream_t stream) {
  auto policy = rmm::exec_policy(stream);
  // count the number of child nodes
  auto num_child_nodes =
      thrust::reduce(policy->on(stream), quad_child_count.begin(),
                     quad_child_count.begin() + num_parent_nodes);

  cudf::size_type num_valid_nodes{0};
  cudf::size_type num_invalid_parent_nodes{0};

  // prune quadrants with fewer points than required
  std::tie(num_invalid_parent_nodes, num_valid_nodes) =
      remove_unqualified_quads(quad_keys, quad_point_count, quad_child_count,
                               quad_level, num_parent_nodes, num_child_nodes,
                               min_size, level_1_size, stream);

  num_parent_nodes -= num_invalid_parent_nodes;

  // construct indicator output column
  // line 6 and 7 of algorithm in Fig. 5 in ref.
  auto indicator =
      construct_non_leaf_indicator(quad_point_count, num_parent_nodes,
                                   num_valid_nodes, min_size, mr, stream);

  // Construct the f_pos output column
  auto f_pos = [&]() {
    // line 8 of algorithm in Fig. 5 in ref.
    // revision to line 8: adjust quad_point_pos based on last-level z-order
    // code
    auto quad_point_pos = compute_flattened_first_point_positions(
        quad_keys, quad_level, quad_point_count, *indicator, num_valid_nodes,
        num_levels, stream);

    // line 9 and 10 of algorithm in Fig. 5 in ref.
    rmm::device_vector<uint32_t> quad_child_pos(num_valid_nodes);
    // line 9 of algorithm in Fig. 5 in ref.
    thrust::replace_if(policy->on(stream), quad_child_count.begin(),
                       quad_child_count.begin() + num_valid_nodes,
                       indicator->view().begin<int8_t>(),
                       !thrust::placeholders::_1, 0);

    // line 10 of algorithm in Fig. 5 in ref.
    thrust::exclusive_scan(policy->on(stream), quad_child_count.begin(),
                           quad_child_count.end(), quad_child_pos.begin(),
                           level_1_size);

    // shrink intermediate device allocation
    shrink_vector(quad_child_pos, num_valid_nodes);

    return copy_if_else<int32_t>(quad_child_pos, quad_point_pos, *indicator,
                                 num_valid_nodes, mr, stream);
  }();

  // Construct the lengths output column
  auto lengths = copy_if_else<int32_t>(quad_child_count, quad_point_count,
                                       *indicator, num_valid_nodes, mr, stream);

  // Construct the keys output column
  auto keys = make_fixed_width_column<int32_t>(num_valid_nodes, stream, mr);

  // Copy quad keys to keys output column
  thrust::copy(policy->on(stream), quad_keys.begin(), quad_keys.end(),
               keys->mutable_view().begin<uint32_t>());

  // Construct the levels output column
  auto levels = make_fixed_width_column<int8_t>(num_valid_nodes, stream, mr);

  // Copy quad levels to levels output column
  thrust::copy(policy->on(stream), quad_level.begin(), quad_level.end(),
               levels->mutable_view().begin<int8_t>());

  std::vector<std::unique_ptr<cudf::column>> cols{};
  cols.reserve(5);
  cols.push_back(std::move(keys));
  cols.push_back(std::move(levels));
  cols.push_back(std::move(indicator));
  cols.push_back(std::move(lengths));
  cols.push_back(std::move(f_pos));
  return std::make_unique<cudf::experimental::table>(std::move(cols));
}

inline std::unique_ptr<cudf::experimental::table> make_empty_quadtree(
    rmm::device_vector<uint32_t> const &quad_keys,
    rmm::device_vector<uint32_t> const &quad_point_count,
    int32_t const num_top_quads, rmm::mr::device_memory_resource *mr,
    hipStream_t stream) {
  auto keys = make_fixed_width_column<int32_t>(num_top_quads, stream, mr);
  auto levels = make_fixed_width_column<int8_t>(num_top_quads, stream, mr);
  auto indicator = make_fixed_width_column<bool>(num_top_quads, stream, mr);
  auto lengths = make_fixed_width_column<int32_t>(num_top_quads, stream, mr);
  auto f_pos = make_fixed_width_column<int32_t>(num_top_quads, stream, mr);

  auto policy = rmm::exec_policy(stream);
  // copy quad keys from the front of the quad_keys list
  thrust::copy(policy->on(stream), quad_keys.begin(),
               quad_keys.begin() + num_top_quads,
               keys->mutable_view().begin<uint32_t>());

  // copy point counts from the front of the quad_point_count list
  thrust::copy(policy->on(stream), quad_point_count.begin(),
               quad_point_count.begin() + num_top_quads,
               lengths->mutable_view().begin<uint32_t>());

  // All leaves are children of the root node (level 0)
  thrust::fill(policy->on(stream), levels->mutable_view().begin<int8_t>(),
               levels->mutable_view().end<int8_t>(), 0);

  // Quad node indicators are false for leaf nodes
  thrust::fill(policy->on(stream), indicator->mutable_view().begin<bool>(),
               indicator->mutable_view().end<bool>(), false);

  // compute f_pos offsets from sizes
  thrust::exclusive_scan(policy->on(stream), lengths->view().begin<uint32_t>(),
                         lengths->view().end<uint32_t>(),
                         f_pos->mutable_view().begin<uint32_t>());

  std::vector<std::unique_ptr<cudf::column>> cols{};
  cols.reserve(5);
  cols.push_back(std::move(keys));
  cols.push_back(std::move(levels));
  cols.push_back(std::move(indicator));
  cols.push_back(std::move(lengths));
  cols.push_back(std::move(f_pos));
  return std::make_unique<cudf::experimental::table>(std::move(cols));
}

template <typename T>
inline std::unique_ptr<cudf::experimental::table> construct_quad_tree(
    cudf::mutable_column_view &x, cudf::mutable_column_view &y, double const x1,
    double const y1, double const x2, double const y2, double const scale,
    cudf::size_type const num_levels, cudf::size_type const min_size,
    rmm::mr::device_memory_resource *mr, hipStream_t stream) {
  // Compute z-order for each point
  auto point_keys = compute_z_keys<T>(
      x, y, stream,
      [x1, y1, x2, y2, num_levels, scale] __device__(auto const &point) {
        T x, y;
        thrust::tie(x, y) = point;
        if (x < x1 || x > x2 || y < y1 || y > y2) {
          // If the point is outside the bbox, return a max_level key
          return static_cast<uint32_t>((1 << (2 * num_levels)) - 1);
        }
        return z_order((x - x1) / scale, (y - y1) / scale);
      });

  rmm::device_vector<uint32_t> quad_keys(x.size());
  rmm::device_vector<uint32_t> quad_point_count(x.size());
  rmm::device_vector<uint32_t> quad_child_count(x.size());

  auto const num_top_quads = compute_full_quads(
      point_keys->view().template begin<uint32_t>(),
      point_keys->view().template end<uint32_t>(),
      thrust::make_constant_iterator<uint32_t>(1), quad_keys.begin(),
      quad_point_count.begin(), thrust::plus<uint32_t>(), stream);

  std::vector<uint32_t> b_pos{};
  std::vector<uint32_t> e_pos{};
  cudf::size_type quad_tree_size{};
  cudf::size_type num_parent_nodes{};

  // compute "full" quads for the tree at each level
  std::tie(num_parent_nodes, quad_tree_size, b_pos, e_pos) =
      compute_full_levels(num_levels, num_top_quads, quad_keys.begin(),
                          quad_point_count.begin(), quad_child_count.begin(),
                          stream);

  // Shrink vectors' underlying device allocations to reduce peak memory usage
  shrink_vector(quad_keys, quad_tree_size);
  shrink_vector(quad_point_count, quad_tree_size);
  shrink_vector(quad_child_count, quad_tree_size);

  // Optimization: can return early if the top level nodes are all leaves
  if (num_parent_nodes <= 0) {
    return make_empty_quadtree(quad_keys, quad_point_count, num_top_quads, mr,
                               stream);
  }

  rmm::device_vector<uint32_t> quad_keys_f{};
  rmm::device_vector<uint32_t> quad_point_count_f{};
  rmm::device_vector<uint32_t> quad_child_count_f{};
  rmm::device_vector<int8_t> quad_level_f{};

  // Reverse the quadtree nodes for easier manipulation (skips the root node)
  std::tie(quad_keys_f, quad_point_count_f, quad_child_count_f, quad_level_f) =
      reverse_tree_levels(quad_keys, quad_point_count, quad_child_count, b_pos,
                          e_pos, num_levels, stream);

  return make_full_quadtree(quad_keys_f, quad_point_count_f, quad_child_count_f,
                            quad_level_f, num_parent_nodes, quad_tree_size,
                            num_levels, min_size, e_pos[0] - b_pos[0], mr,
                            stream);
}

struct dispatch_construct_quadtree {
  template <typename T,
            std::enable_if_t<std::is_floating_point<T>::value> * = nullptr>
  inline std::unique_ptr<cudf::experimental::table> operator()(
      cudf::mutable_column_view &x, cudf::mutable_column_view &y,
      double const x1, double const y1, double const x2, double const y2,
      double const scale, int32_t const num_level, int32_t const min_size,
      rmm::mr::device_memory_resource *mr, hipStream_t stream) {
    return construct_quad_tree<T>(x, y, x1, y1, x2, y2, scale, num_level,
                                  min_size, mr, stream);
  }

  template <typename T,
            std::enable_if_t<!std::is_floating_point<T>::value> * = nullptr>
  inline std::unique_ptr<cudf::experimental::table> operator()(
      cudf::mutable_column_view &x, cudf::mutable_column_view &y,
      double const x1, double const y1, double const x2, double const y2,
      double const scale, int32_t const num_level, int32_t const min_size,
      rmm::mr::device_memory_resource *mr, hipStream_t stream) {
    CUDF_FAIL("Non-floating point operation is not supported");
  }
};

}  // namespace

namespace cuspatial {

std::unique_ptr<cudf::experimental::table> quadtree_on_points(
    cudf::mutable_column_view x, cudf::mutable_column_view y, double const x1,
    double const y1, double const x2, double const y2, double const scale,
    int32_t const num_levels, int32_t const min_size,
    rmm::mr::device_memory_resource *mr) {
  CUSPATIAL_EXPECTS(x.size() == y.size(),
                    "x and y columns might have the same length");
  CUSPATIAL_EXPECTS(x.size() > 0, "point dataset can not be empty");
  CUSPATIAL_EXPECTS(x1 < x2 && y1 < y2, "invalid bounding box (x1,y1,x2,y2)");
  CUSPATIAL_EXPECTS(scale > 0, "scale must be positive");
  CUSPATIAL_EXPECTS(num_levels >= 0 && num_levels < 16,
                    "maximum of levels might be in [0,16)");
  CUSPATIAL_EXPECTS(
      min_size > 0,
      "minimum number of points for a non-leaf node must be larger than zero");

  // detail::quadtree_on_points(x, y, x1, y1, x2, y2, scale, num_levels,
  // min_size, mr, 0);

  return cudf::experimental::type_dispatcher(
      x.type(), dispatch_construct_quadtree{}, x, y, x1, y1, x2, y2, scale,
      num_levels, min_size, mr, hipStream_t{0});
}

}  // namespace cuspatial
