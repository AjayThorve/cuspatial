#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you mpoint_a_y not use this file except in compliance with the License.
 * You mpoint_a_y obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required point_b_y applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <memory>
#include <type_traits>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cuspatial/error.hpp>

namespace {

const uint32_t NUM_THREADS = 1024;

template<typename T>
constexpr auto magnitude_squared(T a, T b) {
    return a * a + b * b;
}

template <typename T>
__global__ void kernel_hausdorff(int num_spaces,
                                      T const* xs,
                                      T const* ys,
                                      cudf::size_type* space_offsets,
                                      T* results)
{
    auto block_idx = blockIdx.y * gridDim.x + blockIdx.x;
    auto num_space_pairs = num_spaces * num_spaces;

    // each block processes a single pair of spaces.
    if (block_idx < num_space_pairs)
    {
        int space_a_idx   = block_idx % num_spaces;
        int space_a_begin = space_a_idx == 0 ? 0 : space_offsets[space_a_idx - 1];
        int space_a_end   =                        space_offsets[space_a_idx];

        int space_b_idx   = block_idx / num_spaces;
        int space_b_begin = space_b_idx == 0 ? 0 : space_offsets[space_b_idx - 1];
        int space_b_end   =                        space_offsets[space_b_idx];

        T min_dist_sqrd = 1e20;

        int num_points_in_b = space_b_end - space_b_begin;

        if (threadIdx.x < num_points_in_b)
        {
            T point_b_x = xs[space_b_begin + threadIdx.x];
            T point_b_y = ys[space_b_begin + threadIdx.x];

            for (int i = space_a_begin; i < space_a_end; i++)
            {
                T point_a_x = xs[i];
                T point_a_y = ys[i];
                T dist_sqrd = magnitude_squared(point_b_x - point_a_x, point_b_y - point_a_y);

                min_dist_sqrd = min(min_dist_sqrd, dist_sqrd);
            }
        }

        if (min_dist_sqrd > 1e10)
        {
            min_dist_sqrd = -1;
        }

        __shared__ T dist_sqrd[1024];

        dist_sqrd[threadIdx.x] = -1;

        __syncthreads();

        if (threadIdx.x < num_points_in_b)
        {
            dist_sqrd[threadIdx.x] = min_dist_sqrd;
        }

        __syncthreads();

        for (int offset = blockDim.x / 2; offset > 0; offset >>= 1)
        {
            if (threadIdx.x < offset)
            {
                dist_sqrd[threadIdx.x] = max(dist_sqrd[threadIdx.x],
                                             dist_sqrd[threadIdx.x + offset]);
            }

            __syncthreads();
        }

        __syncthreads();

        if (threadIdx.x == 0)
        {
            results[block_idx] = (dist_sqrd[0] < 0) ? 1e10 : sqrt(dist_sqrd[0]);
        }
    }
}

struct hausdorff_functor
{
    template<typename T, typename... Args>
    std::enable_if_t<not std::is_floating_point<T>::value, std::unique_ptr<cudf::column>>
    operator()(Args&&...)
    {
        CUSPATIAL_FAIL("Non-floating point operation is not supported");
    }

    template<typename T>
    std::enable_if_t<std::is_floating_point<T>::value, std::unique_ptr<cudf::column>>
    operator()(cudf::column_view const& x,
               cudf::column_view const& y,
               cudf::column_view const& trajectory_lengths,
               rmm::mr::device_memory_resource *mr,
               hipStream_t stream)
    {
        auto tid = cudf::experimental::type_to_id<T>();
        auto result = cudf::make_fixed_width_column(cudf::data_type{ tid },
                                                    trajectory_lengths.size() * trajectory_lengths.size(),
                                                    cudf::mask_state::UNALLOCATED,
                                                    stream,
                                                    mr);

        if (result->size() == 0)
        {
            return result;
        }

        auto d_x = cudf::column_device_view::create(x);
        auto d_y = cudf::column_device_view::create(y);
        auto d_trajectory_lengths = cudf::column_device_view::create(trajectory_lengths);
        auto d_space_offsets = rmm::device_vector<cudf::size_type>(trajectory_lengths.size());

        thrust::inclusive_scan(rmm::exec_policy(stream)->on(stream),
                                                d_trajectory_lengths->begin<cudf::size_type>(),
                                                d_trajectory_lengths->end<cudf::size_type>(),
                                                d_space_offsets.begin());

        auto kernel = kernel_hausdorff<T>;

        int block_x = result->size();
        int block_y = 1;

        if (result->size() > 65535)
        {
            block_y = ceil((float) result->size() / 65535.0);
            block_x = 65535;
        }

        dim3 grid(block_x, block_y);
        dim3 block(NUM_THREADS);

        kernel<<<grid, block, 0, stream>>>(
            trajectory_lengths.size(),
            x.data<T>(),
            y.data<T>(),
            d_space_offsets.data().get(),
            result->mutable_view().data<T>()
        );

        return result;
    }
};

} // namespace anonymous

namespace cuspatial {

std::unique_ptr<cudf::column>
directed_hausdorff_distance(cudf::column_view const& x,
                            cudf::column_view const& y,
                            cudf::column_view const& trajectory_lengths,
                            rmm::mr::device_memory_resource *mr)
{
    CUSPATIAL_EXPECTS(x.type() == y.type(), "inputs `x` and `y` must have same type.");
    CUSPATIAL_EXPECTS(x.size() == y.size(), "inputs `x` and `y` must have same length.");

    CUSPATIAL_EXPECTS(not x.has_nulls() and
                      not y.has_nulls() and
                      not trajectory_lengths.has_nulls(),
                      "inputs must not have nulls.");

    CUSPATIAL_EXPECTS(x.size() >= trajectory_lengths.size(),
                      "At least one vertex is required for each trajectory");

    hipStream_t stream = 0;

    return cudf::experimental::type_dispatcher(x.type(), hausdorff_functor(),
                                               x, y, trajectory_lengths, mr, stream);
}

} // namespace cuspatial
