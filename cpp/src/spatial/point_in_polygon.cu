#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <sys/time.h>
#include <time.h>

#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <utilities/cuda_utils.hpp>
#include <type_traits>
#include <thrust/device_vector.h>
#include <utility/utility.hpp>
#include <cuspatial/point_in_polygon.hpp>

template <typename T>
__global__ void pip_kernel(gdf_size_type pnt_size,const T* const __restrict__ pnt_x,const T* const __restrict__ pnt_y,
        gdf_size_type ply_size,const uint32_t* const __restrict__ ply_fpos,const uint32_t* const __restrict__ ply_rpos,	
        const T* const __restrict__ ply_x,const T* const __restrict__ ply_y,
        uint32_t* const __restrict__ res_bm)
{
    uint32_t mask=0;
    //assuming 1D grid/block config
    uint32_t idx =blockIdx.x*blockDim.x+threadIdx.x;
    if(idx>=pnt_size) return;
    
    T x = pnt_x[idx];
    T y = pnt_y[idx];
    for (uint32_t j = 0; j < ply_size; j++) //for each polygon
    {
       uint32_t r_f = (0 == j) ? 0 : ply_fpos[j-1];
       uint32_t r_t=ply_fpos[j];
       bool in_polygon = false;
       for (uint32_t k = r_f; k < r_t; k++) //for each ring
       {
           uint32_t m = (k==0)?0:ply_rpos[k-1];
           
           /*if(idx==0)
           	printf("%d %d %d %d %d %d\n",j,k,r_f,r_t,m,ply_rpos[k]-1);
           __syncthreads();*/
           	
           for (;m < ply_rpos[k]-1; m++) //for each line segment
           {
              T x0, x1, y0, y1;
              x0 = ply_x[m];
              y0 = ply_y[m];
              x1 = ply_x[m+1];
              y1 = ply_y[m+1];
              
              /*if(idx==0)
              	printf("idx=%3d: %3d %3d %3d %15.10f %15.10f %15.10f %15.10f %15.10f %15.10f \n",idx,j,k,m,x,y,x0,y0,x1,y1);
               __syncthreads();*/
               
              if ((((y0 <= y) && (y < y1)) ||
                   ((y1 <= y) && (y < y0))) &&
                       (x < (x1 - x0) * (y - y0) / (y1 - y0) + x0))
                 in_polygon = !in_polygon;
            }
      }
      if(in_polygon)
      	mask|=(0x01<<j);
   }
   res_bm[idx]=mask;
   //printf("idx=%3d: %08x\n",idx,mask);

}

struct pip_functor {
    template <typename col_type>
    static constexpr bool is_supported()
    {
        return std::is_floating_point<col_type>::value;
    }

    template <typename col_type, std::enable_if_t< is_supported<col_type>() >* = nullptr>
    gdf_column operator()(gdf_column const & pnt_x,gdf_column const & pnt_y,
 			  gdf_column const & ply_fpos,gdf_column const & ply_rpos,
			  gdf_column const & ply_x,gdf_column const & ply_y)
    {
        gdf_column res_bm;
        uint32_t* data;
        
        /*cout<<"output coordinate in pip.cu<<std::endl;
        thrust::device_ptr<col_type> d_pntx_ptr=thrust::device_pointer_cast(static_cast<col_type*>(pnt_x.data));
        thrust::copy(d_pntx_ptr,d_pntx_ptr+pnt_x.size,std::ostream_iterator<col_type>(std::cout, " "));std::cout<<std::endl;  
        thrust::device_ptr<col_type> d_pnty_ptr=thrust::device_pointer_cast(static_cast<col_type*>(pnt_y.data));
        thrust::copy(d_pnty_ptr,d_pnty_ptr+pnt_y.size,std::ostream_iterator<col_type>(std::cout, " "));std::cout<<std::endl;  

        thrust::device_ptr<uint32_t> d_fpos_ptr=thrust::device_pointer_cast(static_cast<uint32_t*>(ply_fpos.data));
        thrust::copy(d_fpos_ptr,d_fpos_ptr+ply_fpos.size,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;       
        thrust::device_ptr<uint32_t> d_rpos_ptr=thrust::device_pointer_cast(static_cast<uint32_t*>(ply_rpos.data));
        thrust::copy(d_rpos_ptr,d_rpos_ptr+ply_rpos.size,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;       
    	
    	thrust::device_ptr<col_type> d_plyx_ptr=thrust::device_pointer_cast(static_cast<col_type*>(ply_x.data));
        thrust::copy(d_plyx_ptr,d_plyx_ptr+ply_x.size,std::ostream_iterator<col_type>(std::cout, " "));std::cout<<std::endl;  
        thrust::device_ptr<col_type> d_plyy_ptr=thrust::device_pointer_cast(static_cast<col_type*>(ply_y.data));
        thrust::copy(d_plyy_ptr,d_plyy_ptr+ply_y.size,std::ostream_iterator<col_type>(std::cout, " "));std::cout<<std::endl;*/       
        
        RMM_TRY( RMM_ALLOC(&data, pnt_y.size * sizeof(uint32_t), 0) );
        gdf_column_view(&res_bm, data, nullptr, pnt_y.size, GDF_INT32);

        struct timeval t0,t1;
        gettimeofday(&t0, nullptr);
        
        gdf_size_type min_grid_size = 0, block_size = 0;
        CUDA_TRY( hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, pip_kernel<col_type>) );
        cudf::util::cuda::grid_config_1d grid{pnt_y.size, block_size, 1};
        
        std::cout<<"pnt_x.size="<<pnt_x.size<<" poly_size="<<ply_fpos.size<<" block_size="<<block_size<<std::endl;
        
        pip_kernel<col_type> <<< grid.num_blocks, block_size >>> (pnt_x.size,
               	static_cast<col_type*>(pnt_x.data), static_cast<col_type*>(pnt_y.data),
        	ply_fpos.size,static_cast<uint32_t*>(ply_fpos.data),static_cast<uint32_t*>(ply_rpos.data),
        	static_cast<col_type*>(ply_x.data), static_cast<col_type*>(ply_y.data),
                static_cast<uint32_t*>(res_bm.data) );
        CUDA_TRY( hipDeviceSynchronize() );

        gettimeofday(&t1, nullptr);	
        float pip_kernel_time = cuspatial::calc_time("pip_kernel_time in ms=",t0,t1);
        
        /*thrust::device_ptr<uint32_t> d_resbm_ptr=thrust::device_pointer_cast(static_cast<uint32_t*>(res_bm.data));
        thrust::copy(d_resbm_ptr,d_resbm_ptr+pnt_x.size,std::ostream_iterator<uint32_t>(std::cout, " "));std::cout<<std::endl;*/       
 
        return res_bm;
    }

    template <typename col_type, std::enable_if_t< !is_supported<col_type>() >* = nullptr>
    gdf_column operator()(gdf_column const & pnt_x,gdf_column const & pnt_y,
 			  gdf_column const & ply_fpos,gdf_column const & ply_rpos,
			  gdf_column const & ply_x,gdf_column const & ply_y)
			  
    {
        CUDF_FAIL("Non-floating point operation is not supported");
    }
};

namespace cuspatial {

/*
 * Point-in-Polygon (PIP) tests among a column of points and a column of
 * polygons. See point_in_polygon.hpp
 */
gdf_column point_in_polygon_bitmap(const gdf_column& points_x,
                                   const gdf_column& points_y,
                                   const gdf_column& poly_fpos,
                                   const gdf_column& poly_rpos,
                                   const gdf_column& poly_x,
                                   const gdf_column& poly_y)
{       
    struct timeval t0,t1;
    gettimeofday(&t0, nullptr);

    CUDF_EXPECTS(points_y.data != nullptr && points_x.data != nullptr, "query point data cannot be empty");
    CUDF_EXPECTS(points_y.dtype == points_x.dtype, "polygon vertex and point data type mismatch for x array ");

    //future versions might allow pnt_(x/y) have null_count>0, which might be useful for taking query results as inputs 
    CUDF_EXPECTS(points_x.null_count == 0 && points_y.null_count == 0, "this version does not support points_x/points_y contains nulls");

    CUDF_EXPECTS(poly_fpos.data != nullptr &&poly_rpos.data!=nullptr, "polygon index cannot be empty");
    CUDF_EXPECTS(poly_fpos.size >0 && (size_t)poly_fpos.size<=sizeof(uint32_t)*8, "#polygon of polygons can not exceed bitmap capacity (32 for unsigned int)");
    CUDF_EXPECTS(poly_y.data != nullptr && poly_x.data != nullptr, "polygon data cannot be empty");
    CUDF_EXPECTS(poly_fpos.size <=poly_rpos.size,"#of polygons must be equal or less than # of rings (one polygon has at least one ring");
    CUDF_EXPECTS(poly_y.size == poly_x.size, "polygon vertice sizes mismatch between x/y arrays");
    CUDF_EXPECTS(points_y.size == points_x.size, "query points size mismatch from between x/y arrays");
    CUDF_EXPECTS(poly_y.dtype == poly_x.dtype, "polygon vertex data type mismatch between x/y arrays");
    CUDF_EXPECTS(poly_y.dtype == points_y.dtype, "polygon vertex and point data type mismatch for y array");
    CUDF_EXPECTS(poly_x.null_count == 0 && poly_y.null_count == 0, "polygon should not contain nulls");

    gdf_column res_bm = cudf::type_dispatcher(points_x.dtype, pip_functor(), 
                                              points_x, points_y, poly_fpos,
                                              poly_rpos,poly_x,poly_y);

    gettimeofday(&t1, nullptr);
    float pip_end2end_time=calc_time("C++ pip_bm end-to-end time in ms=",t0,t1);
    return res_bm;
  }//pip 
  
}// namespace cuspatial
